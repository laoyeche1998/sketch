#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks_1.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 1025;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
  
    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    }
 
    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    
    CUDA_CHECK( hipMalloc((void**)&dA, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dB, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dC, sizeof(double)*N) );
    CUDA_CHECK( hipMemcpy((void*)dA, (void*)hA, sizeof(double)*N, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy((void*)dB, (void*)hB, sizeof(double)*N, hipMemcpyHostToDevice) );
    //#error Add the remaining memory allocations and copies

    // Note the maximum size of threads in a block
    int blockSize = ThreadsInBlock;
    int numBlocks = (N + blockSize - 1) / blockSize;
    dim3 grid(numBlocks), threads(blockSize);

    //// Add the kernel call here
    //CUDA_CHECK( (vector_add<<<grid,threads>>>(dC,dA,dB,N)) );
    vector_add<<<grid,threads>>>(dC,dA,dB,N);
    //#error Add the CUDA kernel call


    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    //hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    CUDA_CHECK( hipMemcpy((void*)hC,(void*)dC,sizeof(double)*N,hipMemcpyDeviceToHost ) );
    CUDA_CHECK( hipFree(dA) );
    CUDA_CHECK( hipFree(dB) );
    CUDA_CHECK( hipFree(dC) );
    //#error Copy back the results and free the allocated memory

    for(int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}